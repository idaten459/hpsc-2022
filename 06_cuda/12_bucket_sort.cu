
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
//#include <vector>

__global__ void bucket_init(int *bucket, int* key, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i>=n) return;
  // 初期化
  bucket[i] = 0;
}

__global__ void bucket_count(int *bucket, int *key, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=n) return;
  // atomicにバケットでカウント
  atomicAdd(&bucket[key[i]],1);
}

__global__ void bucket_key(int *bucket, int *key, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=n) return;
  // sort済列のiの開始indexを計算
  int offset = 0;
  for(int j=0;j<i;j++){
    offset += bucket[j];
  }
  // 全スレッドでoffsetの計算が終了する前に，bucket[i]--が実行されないように同期
  __syncthreads();
  for(; bucket[i]>0; bucket[i]--){
    key[offset++] = i;
  }
}


int main() {
  const int M = 4;
  int n = 50;
  int range = 5;
  int *key;
  int *bucket;
  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  //for (int i=0; i<range; i++) {
  //  bucket[i] = 0;
  //}
  bucket_init<<<(n+M-1)/M,M>>>(bucket, key, n);
  hipDeviceSynchronize();

  //for (int i=0;i<n;i++){
  //  printf("i=%d, bucket[i]=%d\n",i,bucket[i]);
  //for (int i=0; i<n; i++) {
  //  bucket[key[i]]++;
  //}
  bucket_count<<<(n+M-1)/M,M>>>(bucket, key, n);
  hipDeviceSynchronize();

  //for (int i=0, j=0; i<range; i++) {
  //  for (; bucket[i]>0; bucket[i]--) {
  //    key[j++] = i;
  //  }
  //}
  bucket_key<<<(range+M-1)/M,M>>>(bucket, key, range);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(bucket);
  hipFree(key);
}
